
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
using namespace std;
#include <sys/time.h>
double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);
}



__constant__ double kernel[5] = { 0.1200783842,0.2338807566,0.2920817183,0.2338807566,0.1200783842 };

int bmpWidth = 0;
int bmpHeight = 0;
int width;
int lineByte;
unsigned char* pBmpBuf = NULL;
unsigned char* result = NULL;
unsigned char* picture[3] = {NULL,NULL,NULL};
unsigned char* channel[3] = {NULL,NULL,NULL};
pthread_key_t g_key;
typedef struct thread_data {
	long thread_no;
	unsigned char* pBmpBuf = NULL;
	
} thread_data_t;


typedef unsigned int DWORD;
typedef int LONG;
typedef unsigned short WORD;
typedef struct tagBITMAPFILEHEADER {
	WORD    bfType;
	DWORD   bfSize;
	WORD    bfReserved1;
	WORD    bfReserved2;
	DWORD   bfOffBits;
} BITMAPFILEHEADER;
typedef struct tagBITMAPINFOHEADER {
	DWORD      biSize;
	LONG       biWidth;
	LONG       biHeight;
	WORD       biPlanes;
	WORD       biBitCount;
	DWORD      biCompression;
	DWORD      biSizeImage;
	LONG       biXPelsPerMeter;
	LONG       biYPelsPerMeter;
	DWORD      biClrUsed;
	DWORD      biClrImportant;
} BITMAPINFOHEADER;
void read_bmp(unsigned char* &pBmpBuf, int &bmpWidth, int &bmpHeight)
{
	FILE* fp = fopen("timg.bmp", "rb");
	fseek(fp, 14, 0);
	BITMAPINFOHEADER head;
	fread(&head, sizeof(BITMAPINFOHEADER), 1, fp);
	bmpWidth = head.biWidth;
	bmpHeight = head.biHeight;  
	lineByte = (bmpWidth * 3 + 3) / 4 * 4;

	pBmpBuf = new unsigned char[lineByte * bmpHeight];
	//cout << "bmpWidth " << bmpWidth << " bmpHeight " << bmpHeight << endl;
	fread(pBmpBuf, 1, lineByte * bmpHeight, fp);
	fclose(fp);
}

bool saveBmp(char* bmpName, unsigned char* imgBuf, int width, int height)
{
	if (!imgBuf)
		return 0;
	int colorTablesize = 0;
	int lineByte = (width * 24 / 8 + 3) / 4 * 4;
	FILE* fp = fopen(bmpName, "wb");
	if (fp == 0)
		return 0;
	WORD    bfType;
	DWORD   bfSize;
	WORD    bfReserved1;
	WORD    bfReserved2;
	DWORD   bfOffBits;
	bfType = 0x4D42;//bmpÀàÐÍ
	fwrite(&bfType, sizeof(WORD), 1, fp);
	bfSize = 14 + sizeof(BITMAPINFOHEADER) + colorTablesize + lineByte * height;
	fwrite(&bfSize, sizeof(DWORD), 1, fp);
	bfReserved1 = 0;
	fwrite(&bfReserved1, sizeof(WORD), 1, fp);
	bfReserved2 = 0;
	fwrite(&bfReserved2, sizeof(WORD), 1, fp);
	bfOffBits = 54 + colorTablesize;
	fwrite(&bfOffBits, sizeof(DWORD), 1, fp);
	BITMAPINFOHEADER head;
	head.biBitCount = 24;
	head.biClrImportant = 0;
	head.biClrUsed = 0;
	head.biCompression = 0;
	head.biHeight = height;
	head.biPlanes = 1;
	head.biSize = 40;
	head.biSizeImage = lineByte * height;
	head.biWidth = width;
	head.biXPelsPerMeter = 0;
	head.biYPelsPerMeter = 0;
	fwrite(&head, sizeof(BITMAPINFOHEADER), 1, fp);
	fwrite(imgBuf, height * lineByte, 1, fp);
	fclose(fp);
	return 1;

}

__global__ void picture_separation(unsigned char *picture_device, unsigned char *pic_device, int channel, int width, int lineByte, int height){

	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
    int pos = y * lineByte + x *3 + channel;
    int newpos = y * width + x;
    pic_device[newpos] = picture_device[pos];

}
__global__ void picture_conv_row(unsigned char *pic_device, double *pic_middle, int width, int height){

	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
    int pos = y * width + x;
    int begin_pos = y * width;
    int end_pos = (y + 1) * width;
    double myvalue = 0;
    int begin = pos - 2;
    for(int i = begin; i < begin + 5; i++){
    	if(i >= begin_pos && i < end_pos){
    		myvalue += pic_device[i] * kernel[i - begin];
    	}
    }

    int newpos = x * height + y;
    pic_middle[newpos] = myvalue;

}
//矩阵已转置
__global__ void picture_conv_col(double *pic_middle, double * pic_max_pool, int width, int height){
	//height = threadIdx.x
	//width = blockIdx.x
	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
	int pos = x * height + y;
    int begin_pos = x * height;
    int end_pos = (x + 1) * height;
    double myvalue = 0;
    int begin = pos - 2;
    for(int i = begin; i < begin + 5; i++){
    	if(i >= begin_pos && i < end_pos){
    		myvalue += pic_middle[i] * kernel[i - begin];
    	}
    }

    //四个数放一起
    //int newpos = threadIdx.x * width + blockIdx.x;
    //width * 2, height / 2
    
    int x0 = x * 2 + y % 2;
    int y0 = y / 2;//整除
    int newpos = y0 * width * 2 + x0;
    pic_max_pool[newpos] = myvalue;


}
__global__ void max_pooling(double * pic_max_pool, unsigned char * pic_result, int width, int height){
	//width = 4096 * 2
	//height = 2304 / 2
	//blockDim.x = 4096 / 2
	//gridDim.x = 2304 / 2
	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
	int pos = y * width + x;
    double myvalue = pic_max_pool[pos * 4];
    for(int i = pos * 4 + 1; i < pos * 4 + 4; i++){
    	if(myvalue <  pic_max_pool[i]){
    		myvalue = pic_max_pool[i];
    	}
    }
    int newpos = y * width + x;
    pic_result[newpos] = (unsigned char)myvalue;
}
__global__ void picture_combination(unsigned char * pic_result, unsigned char * picture_result, int channel, int width, int lineByte, int height){

	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
    int pos = y * lineByte + x *3 + channel;
    int newpos = y * width + x;
    picture_result[pos] = pic_result[newpos];
}
int main(int argc,char* argv[])
{
	/*
	//定义一个cuda的设备属性结构体
    cudaDeviceProp prop;
    //获取第1个gpu设备的属性信息
    cudaGetDeviceProperties(&prop,0);
    //每个block的最大线程数
    std::cout<<"maxThreadsPerBlock: "<<prop.maxThreadsPerBlock<<std::endl;
    //block的维度
    for(int i=0;i<3;++i) std::cout<<"maxThreadsDim["<<i<<"]: "<<prop.maxThreadsDim[i]<<std::endl;
    //输出最大的gridSize
    std::cout<<std::endl;
    for(int i=0;i<3;++i) std::cout<<"maxGridSize["<<i<<"]: "<<prop.maxGridSize[i]<<std::endl;
	*/

	clock_t start_host = clock();
	read_bmp(pBmpBuf, bmpWidth, bmpHeight);
	cout<<"图片读取时间="<<(double)(clock() - start_host)/1000<<"ms"<<endl;
	width = lineByte / 3;
	result = new unsigned char[lineByte * bmpHeight / 4];
	unsigned char *picture_device = NULL;
	unsigned char *pic_device[3] = {NULL, NULL, NULL};
	double *pic_middle[3] = {NULL, NULL, NULL};
	double *pic_max_pool[3] = {NULL, NULL, NULL};
	unsigned char *pic_result[3] = {NULL, NULL, NULL};
	unsigned char *picture_result = NULL;

  	double iStart,iElaps;
  	iStart=cpuSecond();
  	//图片出入GPU
	hipMalloc((void**)&picture_device,sizeof(unsigned char) * lineByte * bmpHeight);
	hipMemcpy(picture_device, pBmpBuf,sizeof(unsigned char) * lineByte * bmpHeight,hipMemcpyHostToDevice);

	iElaps=cpuSecond()-iStart;
  	cout<<"图片传入GPU时间:" << iElaps*1000<<endl;
	iStart=cpuSecond();
  	//RGB通道分离
	for(int i = 0; i < 3; i++){
		hipMalloc((void**)&pic_device[i],sizeof(unsigned char) * width * bmpHeight);
    	dim3 gridsize(bmpHeight,4,1);
    	dim3 blocksize(1024,1,1);
    	picture_separation<<<gridsize,blocksize>>>(picture_device, pic_device[i], i, width, lineByte, bmpHeight);
			
	}
	//cudaFree(picture_device);	
	//行卷积
	for(int i = 0; i < 3; i++){
		
		hipMalloc((void**)&pic_middle[i],sizeof(double) * width * bmpHeight);
    	dim3 gridsize(bmpHeight,4,1);
    	dim3 blocksize(1024,1,1);
    	picture_conv_row<<<gridsize,blocksize>>>(pic_device[i], pic_middle[i], width, bmpHeight);		
		
		//cudaFree(pic_device[i]);
	}

	//列卷积
	for(int i = 0; i < 3; i++){
    	hipMalloc((void**)&pic_max_pool[i],sizeof(double) * width * bmpHeight);
		dim3 gridsize(bmpHeight,4,1);
    	dim3 blocksize(1024,1,1);

    	picture_conv_col<<<gridsize,blocksize>>>(pic_middle[i], pic_max_pool[i], width, bmpHeight);
		//cudaFree(pic_middle[i]);
	}

	//max pool
	for(int i = 0; i < 3; i++){
		hipMalloc((void**)&pic_result[i],sizeof(unsigned char) * width * bmpHeight / 4);
    	
    	dim3 gridsize(bmpHeight / 2,2,1);
    	dim3 blocksize(1024,1,1);
    	max_pooling<<<gridsize,blocksize>>>(pic_max_pool[i], pic_result[i], width / 2, bmpHeight / 2);
    	//cudaFree(pic_max_pool[i]);
		
	}
	//通道合并
	hipMalloc((void**)&picture_result,sizeof(unsigned char) * lineByte * bmpHeight / 4);
    for(int i = 0; i < 3; i++){
		
    	dim3 gridsize(bmpHeight / 2,2,1);
    	dim3 blocksize(1024,1,1);
    	picture_combination<<<gridsize,blocksize>>>(pic_result[i], picture_result, i, width / 2, lineByte / 2, bmpHeight / 2);

		//cudaFree(pic_result[i]);
	}
	iElaps=cpuSecond()-iStart;
  	cout<<"GPU运行时间:" << iElaps*1000<<endl;
	iStart=cpuSecond();
	hipMemcpy(result, picture_result,sizeof(unsigned char) * lineByte* bmpHeight / 4,hipMemcpyDeviceToHost);

	//cudaDeviceSynchronize();
  	iElaps=cpuSecond()-iStart;
  	cout<<"图片从GPU传出时间:"<<iElaps*1000<<endl;
	iStart=cpuSecond();
	char writePath[] = "Cuda_卷积池化_1952934.bmp";
	saveBmp(writePath, result, bmpWidth / 2, bmpHeight / 2); 
  	iElaps=cpuSecond()-iStart;
  	cout<<"图片保存时间:"<<iElaps*1000<<endl;
	cout<<"total time="<<(double)(clock() - start_host)/1000<<"ms"<<endl;
	return 0;
}


