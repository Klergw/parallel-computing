
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
using namespace std;
#include <sys/time.h>
double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);
}



__constant__ double kernel[5] = { 0.1200783842,0.2338807566,0.2920817183,0.2338807566,0.1200783842 };

int bmpWidth = 0;
int bmpHeight = 0;
int width;
int lineByte;
unsigned char* pBmpBuf = NULL;
unsigned char* result = NULL;
unsigned char* picture[3] = {NULL,NULL,NULL};
unsigned char* channel[3] = {NULL,NULL,NULL};
pthread_key_t g_key;
typedef struct thread_data {
	long thread_no;
	unsigned char* pBmpBuf = NULL;
	
} thread_data_t;


typedef unsigned int DWORD;
typedef int LONG;
typedef unsigned short WORD;
typedef struct tagBITMAPFILEHEADER {
	WORD    bfType;
	DWORD   bfSize;
	WORD    bfReserved1;
	WORD    bfReserved2;
	DWORD   bfOffBits;
} BITMAPFILEHEADER;
typedef struct tagBITMAPINFOHEADER {
	DWORD      biSize;
	LONG       biWidth;
	LONG       biHeight;
	WORD       biPlanes;
	WORD       biBitCount;
	DWORD      biCompression;
	DWORD      biSizeImage;
	LONG       biXPelsPerMeter;
	LONG       biYPelsPerMeter;
	DWORD      biClrUsed;
	DWORD      biClrImportant;
} BITMAPINFOHEADER;
void read_bmp(unsigned char* &pBmpBuf, int &bmpWidth, int &bmpHeight)
{
	FILE* fp = fopen("timg.bmp", "rb");
	fseek(fp, 14, 0);
	BITMAPINFOHEADER head;
	fread(&head, sizeof(BITMAPINFOHEADER), 1, fp);
	bmpWidth = head.biWidth;
	bmpHeight = head.biHeight;  
	lineByte = (bmpWidth * 3 + 3) / 4 * 4;

	pBmpBuf = new unsigned char[lineByte * bmpHeight];
	//cout << "bmpWidth " << bmpWidth << " bmpHeight " << bmpHeight << endl;
	fread(pBmpBuf, 1, lineByte * bmpHeight, fp);
	fclose(fp);
}

bool saveBmp(char* bmpName, unsigned char* imgBuf, int width, int height)
{
	if (!imgBuf)
		return 0;
	int colorTablesize = 0;
	int lineByte = (width * 24 / 8 + 3) / 4 * 4;
	FILE* fp = fopen(bmpName, "wb");
	if (fp == 0)
		return 0;
	WORD    bfType;
	DWORD   bfSize;
	WORD    bfReserved1;
	WORD    bfReserved2;
	DWORD   bfOffBits;
	bfType = 0x4D42;//bmp????????
	fwrite(&bfType, sizeof(WORD), 1, fp);
	bfSize = 14 + sizeof(BITMAPINFOHEADER) + colorTablesize + lineByte * height;
	fwrite(&bfSize, sizeof(DWORD), 1, fp);
	bfReserved1 = 0;
	fwrite(&bfReserved1, sizeof(WORD), 1, fp);
	bfReserved2 = 0;
	fwrite(&bfReserved2, sizeof(WORD), 1, fp);
	bfOffBits = 54 + colorTablesize;
	fwrite(&bfOffBits, sizeof(DWORD), 1, fp);
	BITMAPINFOHEADER head;
	head.biBitCount = 24;
	head.biClrImportant = 0;
	head.biClrUsed = 0;
	head.biCompression = 0;
	head.biHeight = height;
	head.biPlanes = 1;
	head.biSize = 40;
	head.biSizeImage = lineByte * height;
	head.biWidth = width;
	head.biXPelsPerMeter = 0;
	head.biYPelsPerMeter = 0;
	fwrite(&head, sizeof(BITMAPINFOHEADER), 1, fp);
	fwrite(imgBuf, height * lineByte, 1, fp);
	fclose(fp);
	return 1;

}

__global__ void picture_separation(unsigned char *picture_device, unsigned char *pic_device, int channel, int width, int lineByte, int height){

	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
    int pos = y * lineByte + x *3 + channel;
    int newpos = y * width + x;
    pic_device[newpos] = picture_device[pos];

}
__global__ void picture_conv_row(unsigned char *pic_device, double *pic_middle, int width, int height){

	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
    int pos = y * width + x;
    int begin_pos = y * width;
    int end_pos = (y + 1) * width;
    double myvalue = 0;
    int begin = pos - 2;
    for(int i = begin; i < begin + 5; i++){
    	if(i >= begin_pos && i < end_pos){
    		myvalue += pic_device[i] * kernel[i - begin];
    	}
    }

    int newpos = x * height + y;
    pic_middle[newpos] = myvalue;

}
//???????????????
__global__ void picture_conv_col(double *pic_middle, double * pic_max_pool, int width, int height){
	//height = threadIdx.x
	//width = blockIdx.x
	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
	int pos = x * height + y;
    int begin_pos = x * height;
    int end_pos = (x + 1) * height;
    double myvalue = 0;
    int begin = pos - 2;
    for(int i = begin; i < begin + 5; i++){
    	if(i >= begin_pos && i < end_pos){
    		myvalue += pic_middle[i] * kernel[i - begin];
    	}
    }

    //??????????????????
    //int newpos = threadIdx.x * width + blockIdx.x;
    //width * 2, height / 2
    
    int x0 = x * 2 + y % 2;
    int y0 = y / 2;//??????
    int newpos = y0 * width * 2 + x0;
    pic_max_pool[newpos] = myvalue;


}
__global__ void max_pooling(double * pic_max_pool, unsigned char * pic_result, int width, int height){
	//width = 4096 * 2
	//height = 2304 / 2
	//blockDim.x = 4096 / 2
	//gridDim.x = 2304 / 2
	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
	int pos = y * width + x;
    double myvalue = pic_max_pool[pos * 4];
    for(int i = pos * 4 + 1; i < pos * 4 + 4; i++){
    	if(myvalue <  pic_max_pool[i]){
    		myvalue = pic_max_pool[i];
    	}
    }
    int newpos = y * width + x;
    pic_result[newpos] = (unsigned char)myvalue;
}
__global__ void picture_combination(unsigned char * pic_result, unsigned char * picture_result, int channel, int width, int lineByte, int height){

	int x = blockIdx.y * 1024 + threadIdx.x;
	int y = blockIdx.x;
    int pos = y * lineByte + x *3 + channel;
    int newpos = y * width + x;
    picture_result[pos] = pic_result[newpos];
}
int main(int argc,char* argv[])
{
	/*
	//????????????cuda????????????????????????
    cudaDeviceProp prop;
    //?????????1???gpu?????????????????????
    cudaGetDeviceProperties(&prop,0);
    //??????block??????????????????
    std::cout<<"maxThreadsPerBlock: "<<prop.maxThreadsPerBlock<<std::endl;
    //block?????????
    for(int i=0;i<3;++i) std::cout<<"maxThreadsDim["<<i<<"]: "<<prop.maxThreadsDim[i]<<std::endl;
    //???????????????gridSize
    std::cout<<std::endl;
    for(int i=0;i<3;++i) std::cout<<"maxGridSize["<<i<<"]: "<<prop.maxGridSize[i]<<std::endl;
	*/

	clock_t start_host = clock();
	read_bmp(pBmpBuf, bmpWidth, bmpHeight);
	cout<<"??????????????????="<<(double)(clock() - start_host)/1000<<"ms"<<endl;
	width = lineByte / 3;
	result = new unsigned char[lineByte * bmpHeight / 4];
	unsigned char *picture_device = NULL;
	unsigned char *pic_device[3] = {NULL, NULL, NULL};
	double *pic_middle[3] = {NULL, NULL, NULL};
	double *pic_max_pool[3] = {NULL, NULL, NULL};
	unsigned char *pic_result[3] = {NULL, NULL, NULL};
	unsigned char *picture_result = NULL;

  	double iStart,iElaps;
  	iStart=cpuSecond();
  	//????????????GPU
	hipMalloc((void**)&picture_device,sizeof(unsigned char) * lineByte * bmpHeight);
	hipMemcpy(picture_device, pBmpBuf,sizeof(unsigned char) * lineByte * bmpHeight,hipMemcpyHostToDevice);

	iElaps=cpuSecond()-iStart;
  	cout<<"????????????GPU??????:" << iElaps*1000<<endl;
	iStart=cpuSecond();
  	//RGB????????????
	for(int i = 0; i < 3; i++){
		hipMalloc((void**)&pic_device[i],sizeof(unsigned char) * width * bmpHeight);
    	dim3 gridsize(bmpHeight,4,1);
    	dim3 blocksize(1024,1,1);
    	picture_separation<<<gridsize,blocksize>>>(picture_device, pic_device[i], i, width, lineByte, bmpHeight);
			
	}
	//cudaFree(picture_device);	
	//?????????
	for(int i = 0; i < 3; i++){
		
		hipMalloc((void**)&pic_middle[i],sizeof(double) * width * bmpHeight);
    	dim3 gridsize(bmpHeight,4,1);
    	dim3 blocksize(1024,1,1);
    	picture_conv_row<<<gridsize,blocksize>>>(pic_device[i], pic_middle[i], width, bmpHeight);		
		
		//cudaFree(pic_device[i]);
	}

	//?????????
	for(int i = 0; i < 3; i++){
    	hipMalloc((void**)&pic_max_pool[i],sizeof(double) * width * bmpHeight);
		dim3 gridsize(bmpHeight,4,1);
    	dim3 blocksize(1024,1,1);

    	picture_conv_col<<<gridsize,blocksize>>>(pic_middle[i], pic_max_pool[i], width, bmpHeight);
		//cudaFree(pic_middle[i]);
	}

	//max pool
	for(int i = 0; i < 3; i++){
		hipMalloc((void**)&pic_result[i],sizeof(unsigned char) * width * bmpHeight / 4);
    	
    	dim3 gridsize(bmpHeight / 2,2,1);
    	dim3 blocksize(1024,1,1);
    	max_pooling<<<gridsize,blocksize>>>(pic_max_pool[i], pic_result[i], width / 2, bmpHeight / 2);
    	//cudaFree(pic_max_pool[i]);
		
	}
	//????????????
	hipMalloc((void**)&picture_result,sizeof(unsigned char) * lineByte * bmpHeight / 4);
    for(int i = 0; i < 3; i++){
		
    	dim3 gridsize(bmpHeight / 2,2,1);
    	dim3 blocksize(1024,1,1);
    	picture_combination<<<gridsize,blocksize>>>(pic_result[i], picture_result, i, width / 2, lineByte / 2, bmpHeight / 2);

		//cudaFree(pic_result[i]);
	}
	iElaps=cpuSecond()-iStart;
  	cout<<"GPU????????????:" << iElaps*1000<<endl;
	iStart=cpuSecond();
	hipMemcpy(result, picture_result,sizeof(unsigned char) * lineByte* bmpHeight / 4,hipMemcpyDeviceToHost);

	//cudaDeviceSynchronize();
  	iElaps=cpuSecond()-iStart;
  	cout<<"?????????GPU????????????:"<<iElaps*1000<<endl;
	iStart=cpuSecond();
	char writePath[] = "Cuda_????????????_1952934.bmp";
	saveBmp(writePath, result, bmpWidth / 2, bmpHeight / 2); 
  	iElaps=cpuSecond()-iStart;
  	cout<<"??????????????????:"<<iElaps*1000<<endl;
	cout<<"total time="<<(double)(clock() - start_host)/1000<<"ms"<<endl;
	return 0;
}


